#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int num_threads;
int num_blocks;
int inputSize;
float *values

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

void sampleSort(float *values){
    

}

void correctness_check(){
    //check if sorted
    int i;
    for(i = 0; i < inputSize - 1; i++){
        if(values[i] > values[i+1]){
            printf("Error: Array not sorted\n");
            return;
        }
    }
    printf("Array sorted correctly\n");
    return;
}

void data_init(){
    values = (float*) malloc(inputSize * sizeof(float));
    array_fill(values, inputSize);
}

void main(int argc, char *argv[]){
    num_threads = atoi(argv[1]);
    inputSize = atoi(argv[2]);
    num_blocks = inputSize/num_threads;

    data_init();

    sampleSort(values);

    correctness_check();

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "SampleSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "MPIwithCUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(float)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", inputSize); // The number of elements in input dataset (1000)
    adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_procs", num_procs); // The number of processors (MPI ranks)
    adiak::value("num_threads", num_threads); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", num_blocks); // The number of CUDA blocks 
    adiak::value("group_num", 6); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", implementation_source) // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").
    return;
}

