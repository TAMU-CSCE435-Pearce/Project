#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <iostream>
#include <algorithm>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int THREADS;
int BLOCKS;
int NUM_VALS;

// Cali Regions
const char* main_region = "main";
const char* data_init = "data_init";
const char* comp = "comp";
const char* comm = "comm";
const char* comp_small = "comp_small";
const char* comm_small = "comm_small";
const char* comp_large = "comp_large";
const char* comm_large = "comm_large";
const char* correctness_check = "correctness_check";

using namespace std;

int correctnessCheck(int *arr, int size) {
  CALI_MARK_BEGIN(correctness_check);
  for (int i=0; i<size-1; i++) {
    if (arr[i+1] < arr[i])
      return 0;  // means it's not ordered correctly
  }
  CALI_MARK_END(correctness_check);

  return 1;
}

void dataInit(int *arr, int size, int inputType) {
  int numToSwitch = size / 100;
  int firstIndex, secondIndex;
  switch (inputType) {
    case 1:
      // sorted
      for (int i=0; i<size; i++) {
        arr[i] = i;
      }
      break;
    case 2:
      // reverse sorted
      for (int i=0; i<size; i++) {
        arr[i] = size-i;
      }
      break;
    case 3:
      // randomized
      for (int i=0; i<size; i++) {
        arr[i] = rand() % RAND_MAX;
      }
      break;
    case 4:
      // 1% perturbed
      for (int i=0; i<size; i++) {
        arr[i] = i;
      }
      if (numToSwitch == 0)  // at the very least one value should be switched
        numToSwitch = 1;
      
      for (int i=0; i<numToSwitch; i++) {
        firstIndex = rand() % size;
        secondIndex = rand() % size;
        //printf("first index: %d, second index: %d\n", firstIndex, secondIndex);
        while (firstIndex == secondIndex) {
          secondIndex = rand() % size;
        } 
        std::swap(arr[firstIndex], arr[secondIndex]); 
      }
      break;
    default:
      printf("THAT'S NOT A VALID INPUT TYPE");
      break;
  }
}

void finalSort(int** buckets, int rows) {
    for (int r = 0; r < rows; ++r) {
        for (int i = 0; i < NUM_VALS - 1; ++i) {
          for (int j = 0; j < NUM_VALS - i - 1; ++j) {
              if (buckets[r][j] > buckets[r][j + 1]) {
                  // Swap elements if they are in the wrong order
                  int temp = buckets[r][j];
                  buckets[r][j] = buckets[r][j + 1];
                  buckets[r][j + 1] = temp;
              }
          }
        }
    }
}

void chooseSplitters(int *splitters, int *samples) {
    // samples
    int samplesSize = BLOCKS * (BLOCKS-1);
    for (int i = 0; i < samplesSize - 1; ++i) {
      for (int j = 0; j < samplesSize - i - 1; ++j) {
          if (samples[j] > samples[j + 1]) {
              // Swap elements if they are in the wrong order
              int temp = samples[j];
              samples[j] = samples[j + 1];
              samples[j + 1] = temp;
          }
      }
    }
    
    // choose splitters
    int spacing = std::ceil((float)samplesSize/(float)BLOCKS);
    int splitterIndex = spacing-1;
    
    for (int i = 0; i < BLOCKS-1; i++) {
      splitters[i] = samples[splitterIndex];
      splitterIndex += spacing;
    }
}


__global__ void chooseSamples(int* data, int *samples, int numBlocks) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    // only smallest thread sorts block
    if (threadIdx.x == 0) {
      // sort each block
      for (int i = 0; i < blockDim.x - 1; ++i) {
        for (int j = 0; j < blockDim.x - i - 1; ++j) {
            if (data[index + j] > data[index + j + 1]) {
                // Swap elements if they are in the wrong order
                int temp = data[index + j];
                data[index + j] = data[index + j + 1];
                data[index + j + 1] = temp;
            }
        }
      }
      
      // choose samples from sorted block
      int spacing = blockDim.x /(numBlocks-1);
      int sampleIndex = spacing-1;
      
      for (int i = 0; i < numBlocks-1; i++) {
        samples[blockIdx.x * (numBlocks-1) + i] = data[index+sampleIndex];
        sampleIndex += spacing;
      }
    }

    
}

__global__ void sampleSort(int* data, int** buckets, int* splitters, int* flattenedArr, int numSplitters, int numVals) {
    
    int index = blockDim.x * blockIdx.x + threadIdx.x;
       
    // each thread checks which bucket they fall into
    int j = 0;
    while(j < numSplitters) {  // j being which bucket it should belong to
  			if (j == numSplitters-1) {
          // means it should go in last bucket
          // makes sure that we don't try to access splitters[buckets.size()-1]. will go out of range
          buckets[j][index] = data[index];
          break;
        }
        if(data[index] < splitters[j]) {
  				buckets[j][index] = data[index];
          break;
  			}
  			j++;
    }
    
    // store bucket values in flattened array
    int arrIndex = 0;
    for (int i = 0; i < numSplitters; ++i) {
        for (int j = 0; j < numVals; ++j) {
            flattenedArr[arrIndex++] = buckets[i][j];
        }
    }
}

int main(int argc, char *argv[])
{
    int inputType;
    inputType = atoi(argv[3]);
    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);
    printf("Input type: %d\n", inputType);

    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();

    // host data
    int* hostData = new int[NUM_VALS];
    int *splitters = new int[BLOCKS-1]; 
    int *samples = (int*)malloc(sizeof(int) * (BLOCKS-1)*BLOCKS);  // each block picks out potential splitter candidates
    
    // initialize data according to inputType
    dataInit(hostData, NUM_VALS, inputType);
    
    cout << "original arr" << endl;  
    for (int i = 0; i < NUM_VALS; ++i) {
        cout << hostData[i] << " ";
    }
    cout << endl;  

    // device data
    int* devData, *dsplitters, *dsamples;
    hipMalloc((void**)&devData, NUM_VALS * sizeof(int));
    hipMalloc((void**)&dsplitters, (BLOCKS-1) * sizeof(int));
    hipMalloc((void**)&dsamples, (BLOCKS-1)*BLOCKS * sizeof(int));
    
    // send chunks to device
    hipMemcpy(devData, hostData, NUM_VALS * sizeof(int), hipMemcpyHostToDevice);
    
    // have device sort and send back samples
    chooseSamples<<<BLOCKS, THREADS>>>(devData, dsamples, BLOCKS);
    
    // receive samples from device
    hipMemcpy(samples, dsamples, (BLOCKS-1) * BLOCKS * sizeof(int), hipMemcpyDeviceToHost);

    // sort samples and choose splitters
    chooseSplitters(splitters, samples);
    
    // allocate memory for host and device 2d bucket arrays
    int rows = BLOCKS-1;
    int** buckets = new int*[rows];
    int** dbuckets;
    int* dflattenedArr;
    hipMalloc((void**)&dflattenedArr, rows * NUM_VALS * sizeof(int));
    for (int i = 0; i < rows; ++i) {
        buckets[i] = new int[NUM_VALS];
    }
    
    // initalize buckets with -1 so we know what to remove later
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < NUM_VALS; j++) {
        buckets[i][j] = -1;
      }
    }
    
    // Allocate device memory for the 2D array
    hipMalloc((void**)&dbuckets, rows * sizeof(int*));
    for (int i = 0; i < rows; ++i) {
        int* d_row;
        hipMalloc((void**)&d_row, NUM_VALS * sizeof(int));
        hipMemcpy(d_row, buckets[i], NUM_VALS * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dbuckets + i, &d_row, sizeof(int*), hipMemcpyHostToDevice);
    }
    
    // send chunks to device w/ splitters
    hipMemcpy(dsplitters, splitters, sizeof(int) * (BLOCKS-1), hipMemcpyHostToDevice);
    sampleSort<<<BLOCKS, THREADS>>>(devData, dbuckets, dsplitters, dflattenedArr, BLOCKS-1, NUM_VALS);
    
    int *flattenedArr = (int*)malloc(sizeof(int) * (BLOCKS-1)*NUM_VALS);
    hipMemcpy(flattenedArr, dflattenedArr, (BLOCKS-1) * NUM_VALS * sizeof(int), hipMemcpyDeviceToHost);
    
    // initializing unflattened arr
    int** unflattened = new int*[rows];
    for (int i = 0; i < rows; ++i) {
        unflattened[i] = new int[NUM_VALS];
    }
    
    // unflatten the arr
    int index = 0;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < NUM_VALS; ++j) {
            unflattened[i][j] = flattenedArr[index++];
        }
    }
    
    // final sort each row
    finalSort(unflattened, rows);
    
    // append to one array and done!
    int* finalArr = new int[NUM_VALS];
    int finalArrIndex = 0;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < NUM_VALS; ++j) {
            if (unflattened[i][j] != -1) {
              finalArr[finalArrIndex++] = unflattened[i][j];
            }
        }
        
    }
    
    cout << "FINAL ARRAY" << endl;
    for (int i = 0; i < NUM_VALS; i++) {
      cout << finalArr[i] << " ";
    }
    
    if (correctnessCheck(finalArr, NUM_VALS)) {
      printf("\nCORRECT");
    } else {
      printf("\nINCORRECT");
    }

    // Flush Caliper output before finalizing MPI
    mgr.stop();
    mgr.flush();
};

