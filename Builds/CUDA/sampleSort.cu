#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <iostream>
#include <algorithm>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int THREADS;
int BLOCKS;
int NUM_VALS;

/* Define Caliper region names */
const char* main_region = "main";
const char* data_init = "data_init";
const char* correctness_check = "correctness_check";
const char* comm = "comm";
const char* comm_small = "comm_small";
const char* comm_large = "comm_large";
const char* comp = "comp";
const char* comp_small = "comp_small";
const char* comp_large = "comp_large";

using namespace std;

void chooseSplitters(int *splitters, int *samples) {
    // samples
    int samplesSize = BLOCKS * (BLOCKS-1);
    for (int i = 0; i < samplesSize - 1; ++i) {
      for (int j = 0; j < samplesSize - i - 1; ++j) {
          if (samples[j] > samples[j + 1]) {
              // Swap elements if they are in the wrong order
              int temp = samples[j];
              samples[j] = samples[j + 1];
              samples[j + 1] = temp;
          }
      }
    }
    
    cout << "sorted samples" << endl;
    for (int i = 0; i < samplesSize; ++i) {
        cout << samples[i] << " ";
    }
    cout << endl;
    
    // choose splitters
    int spacing = std::ceil((float)samplesSize/(float)BLOCKS);
    int splitterIndex = spacing-1;
    
    for (int i = 0; i < BLOCKS-1; i++) {
      splitters[i] = samples[splitterIndex];
      splitterIndex += spacing;
    }
}


__global__ void chooseSamples(int* data, int* out, int *samples, int numBlocks) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    // only smallest thread sorts block
    if (threadIdx.x == 0) {
      // sort each block
      for (int i = 0; i < blockDim.x - 1; ++i) {
        for (int j = 0; j < blockDim.x - i - 1; ++j) {
            if (data[index + j] > data[index + j + 1]) {
                // Swap elements if they are in the wrong order
                int temp = data[index + j];
                data[index + j] = data[index + j + 1];
                data[index + j + 1] = temp;
            }
        }
      }
      
      // choose samples from sorted block
      int spacing = blockDim.x /(numBlocks-1);
      int sampleIndex = spacing-1;
      
      for (int i = 0; i < numBlocks-1; i++) {
        samples[blockIdx.x * (numBlocks-1) + i] = data[index+sampleIndex];
        printf("sample index: %d ", blockIdx.x * (numBlocks-1) + i);
        //printf("index: %d\n", index);
        sampleIndex += spacing;
      }
      
      
      // Write the sorted data back to the global memory
      for (int i = 0; i < blockDim.x; ++i) {
          out[blockDim.x * blockIdx.x + i] = data[index + i];
      }
    }

    
}

__global__ void sampleSort(int* data, int** buckets, int* splitters, int* flattenedArr, int numSplitters, int numVals) {
    
    int index = blockDim.x * blockIdx.x + threadIdx.x;
       
    // each thread checks which bucket they fall into
    int j = 0;
    while(j < numSplitters) {  // j being which bucket it should belong to
  			if (j == numSplitters-1) {
          // means it should go in last bucket
          // makes sure that we don't try to access splitters[buckets.size()-1]. will go out of range
          buckets[j][index] = data[index];
          break;
        }
        if(data[index] < splitters[j]) {
  				buckets[j][index] = data[index];
          break;
  			}
  			j++;
    }
    
    // store bucket values in flattened array
    int arrIndex = 0;
    for (int i = 0; i < numSplitters; ++i) {
        for (int j = 0; j < numVals; ++j) {
            flattenedArr[arrIndex++] = buckets[i][j];
        }
    }
}

int main(int argc, char *argv[])
{
    int inputType;
    inputType = atoi(argv[3]);
    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);
    printf("Input type: %d\n", inputType);

    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();

    // host data
    int* hostData = new int[NUM_VALS];
    int *out = (int*)malloc(sizeof(int) * NUM_VALS);
    int *splitters = new int[BLOCKS-1]; 
    int *samples = (int*)malloc(sizeof(int) * (BLOCKS-1)*BLOCKS);  // each block picks out potential splitter candidates
    
    // initialize data according to inputType
    for (int i = 0; i < NUM_VALS; ++i) {
        hostData[i] = NUM_VALS-i;
    }
    cout << "original arr" << endl;  
    for (int i = 0; i < NUM_VALS; ++i) {
        cout << hostData[i] << " ";
    }
    cout << endl;  

    // device data
    int* devData, *dout, *dsplitters, *dsamples;
    hipMalloc((void**)&devData, NUM_VALS * sizeof(int));
    hipMalloc((void**)&dout, NUM_VALS * sizeof(int));
    hipMalloc((void**)&dsplitters, (BLOCKS-1) * sizeof(int));
    hipMalloc((void**)&dsamples, (BLOCKS-1)*BLOCKS * sizeof(int));
    
    // send chunks to device
    hipMemcpy(devData, hostData, NUM_VALS * sizeof(int), hipMemcpyHostToDevice);
    
    // have device sort and send back samples
    chooseSamples<<<BLOCKS, THREADS>>>(devData, dout, dsamples, BLOCKS);
    
    // receive samples from device
    hipMemcpy(out, dout, sizeof(int) * NUM_VALS, hipMemcpyDeviceToHost);
    hipMemcpy(samples, dsamples, (BLOCKS-1) * BLOCKS * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < NUM_VALS; ++i) {
        cout << out[i] << " ";
    }
    cout << endl;
    cout << "SAMPLES" << endl;
    for (int i = 0; i < (BLOCKS-1)*BLOCKS; ++i) {
        cout << samples[i] << " ";
    }
    cout << endl;

    // sort samples and choose splitters
    chooseSplitters(splitters, samples);
    cout << "chosen splitters" << endl;
    for (int i = 0; i < BLOCKS-1; ++i) {
        cout << splitters[i] << " ";
    }
    cout << endl;
    
    // allocate memory for host and device 2d bucket arrays
    int rows = BLOCKS-1;
    int** buckets = new int*[rows];
    int** dbuckets;
    int* dflattenedArr;
    hipMalloc((void**)&dflattenedArr, rows * NUM_VALS * sizeof(int));
    hipMalloc((void**)&dbuckets, rows * sizeof(int*));
    for (int i = 0; i < rows; ++i) {
        hipMalloc((void**)&buckets[i], NUM_VALS * sizeof(int));
        hipMemcpy(dbuckets + i, &(buckets[i]), sizeof(int*), hipMemcpyHostToDevice);
    }
    hipMemcpy(dbuckets, buckets, rows * sizeof(int*), hipMemcpyHostToDevice);  // Copy the row pointers to the device
    
    // send chunks to device w/ splitters
    hipMemcpy(dsplitters, splitters, sizeof(int) * (BLOCKS-1), hipMemcpyHostToDevice);
    sampleSort<<<BLOCKS, THREADS>>>(devData, dbuckets, dsplitters, dflattenedArr, BLOCKS-1, NUM_VALS);
    
    int *flattenedArr = (int*)malloc(sizeof(int) * (BLOCKS-1)*NUM_VALS);
    hipMemcpy(flattenedArr, dflattenedArr, (BLOCKS-1) * NUM_VALS * sizeof(int), hipMemcpyDeviceToHost);
    cout << "flattened" << endl;
    for (int i = 0; i < NUM_VALS * (BLOCKS-1); i++) {
      cout << flattenedArr[i] << " ";
    }
    
    int** unflattened = new int*[rows];
    for (int i = 0; i < rows; ++i) {
        unflattened[i] = new int[NUM_VALS];
    }
    
    // unflatten the arr
    int index = 0;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < NUM_VALS; ++j) {
            unflattened[i][j] = flattenedArr[index++];
        }
    }
    
    cout << "unflattened" << endl;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < NUM_VALS; ++j) {
            cout << unflattened[i][j] << " ";
        }
        cout << endl;
    }
    
    
    
    // hipMemcpy(buckets, dbuckets, rows * sizeof(int*), hipMemcpyDeviceToHost);
    
    /*
    cout << "buckets" << endl;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < NUM_VALS; j++) {
          cout << buckets[i][j] << " ";
        }
    }
    cout << endl;
    */
    
    // send buckets back from device to host and append to global 2d buckets arr
    
    // final sort each row
    
    // append to one array and done!


    // Flush Caliper output before finalizing MPI
    mgr.stop();
    mgr.flush();
};

