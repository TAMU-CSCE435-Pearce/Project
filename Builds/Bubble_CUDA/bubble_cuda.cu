#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <adiak.hpp>

using namespace std;

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* data_init = "data_init";
const char* data_gen_h2d = "data_gen_h2d";
const char* data_gen_d2h = "data_gen_d2h";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* comm = "comm";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";
const char* correctness_check = "correctness_check";
const char* correctness_h2d = "correctness_h2d";
const char* correctness_d2h = "correctness_d2h";


__global__ void random_fill(float* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (float)hiprand_uniform(&state) * size;
}

__global__ void sorted_fill(float* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (float)index;
}

__global__ void reverse_fill(float* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (float)(size - index - 1);
}

__global__ void nearly_fill(float* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (float)hiprand_uniform(&state) * blockIdx.x;
}

__global__ void confirm_sorted_step(float* nums, int size, bool* sorted) {
    // __shared__ bool sorted;
    // sorted = true;
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if(index < size - 1) {
        if(nums[index] > nums[index + 1]) {
            *sorted = false;
        }
    }

    // __syncthreads();

    // // parallel reduction to check if whole array is sorted
    // for(int i = 1; i < blockDim.x; i *= 2) {
    //     if(index % (2 * i) == 0) {
    //         sorted = sorted && __shfl_down_sync(0xFFFFFFFF, sorted, i);
    //     }
    //     __syncthreads();
    // }

    // if(index == 0) {
    //     *isSorted = sorted;
    // }
}

void fill_array(float* nums, const char* input_type) {
    float *dev_nums;
    size_t size = NUM_VALS * sizeof(float);
    
    hipMalloc((void**) &dev_nums, size);

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(data_gen_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    CALI_MARK_END(data_gen_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // FILLING ARRAY
    CALI_MARK_BEGIN(data_init);
    if(strcmp(input_type, "random") == 0) {
        random_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "sorted") == 0) {
        sorted_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "reverse") == 0) {
        reverse_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "nearly") == 0) {
        nearly_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    CALI_MARK_END(data_init);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(data_gen_d2h);
    hipMemcpy(nums, dev_nums, size, hipMemcpyDeviceToHost);
    CALI_MARK_BEGIN(data_gen_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
}

bool confirm_sorted(float* nums) {
    float *dev_nums;
    bool *dev_sorted;
    bool sorted = true;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void**) &dev_nums, size);
    hipMalloc((void**) &dev_sorted, sizeof(bool));

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(correctness_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_sorted, &sorted, sizeof(bool), hipMemcpyHostToDevice);
    CALI_MARK_END(correctness_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // CHECKING CORRECTNESS
    CALI_MARK_BEGIN(correctness_check);
    confirm_sorted_step<<<blocks, threads>>>(dev_nums, NUM_VALS, dev_sorted);
    hipDeviceSynchronize();
    CALI_MARK_END(correctness_check);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(correctness_d2h);
    hipMemcpy(&sorted, dev_sorted, sizeof(bool), hipMemcpyDeviceToHost);
    CALI_MARK_END(correctness_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
    hipFree(dev_sorted);
    return sorted;
}

int main(int argc, char *argv[]) {
    CALI_CXX_MARK_FUNCTION;
    cali::ConfigManager mgr;
    mgr.start();

    // retrieve user input
    const char* input_type = argv[1];
    THREADS = atoi(argv[2]);
    NUM_VALS = atoi(argv[3]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    // initialize array
    float *nums = (float*) malloc(NUM_VALS * sizeof(float));
    
    // fill array
    fill_array(nums, input_type);
    cout << "Data Initialized" << endl;

    // test print array
    // for(int i = 0; i < NUM_VALS; i++) {
    //     cout << nums[i] << " ";
    // }
    // cout << endl;

    // check correctness
    if(confirm_sorted(nums)) {
        cout << "Correctness Check Passed!" << endl;
    }
    else {
        cout << "Correctness Check Failed..." << endl;
    }

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "Odd Even Sort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(float)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", input_type); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_threads", NUM_VALS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    //adiak::value("group_num", group_number); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "AI"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    mgr.stop();
    mgr.flush();

    free(nums);
}
  