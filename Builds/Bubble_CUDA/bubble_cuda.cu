#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <adiak.hpp>

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* data_init = "data_init";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";
const char* correctness_check = "correctness_check";

__global__ void fill_array(float* nums, const char* input_type, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(input_type == "random") {
        nums[index] = rand() % size;
    }
    if(input_type == "sorted") {
        nums[index] = index;
    }
    if(input_type == "reverse") {
        nums[index] = size - index - 1;
    }
}

int main(int argc, char *argv[]) {
    const char* input_type = argv[1];
    THREADS = atoi(argv[2]);
    NUM_VALS = atoi(argv[3]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();

    float* nums;
    hipMalloc((void**)&nums, sizeof(float) * NUM_VALS);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    fill_array<<<blocks, threads>>>(nums, input_type, NUM_VALS);
    // test
    for(float f : nums) {
        cout << f << " ";
    }
    cout << endl;

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", algorithm); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", programmingModel); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(float)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", input_type); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_threads", NUM_VALS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    //adiak::value("group_num", group_number); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Online") // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    mgr.stop();
    mgr.flush();
}
  