#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

int THREADS;
int BLOCKS;
int NUM_VALS;

hipEvent_t start, stop;
const char* bitonic_sort_step_region = "bitonic_sort_step";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";

// Store results in these variables.
float effective_bandwidth_gb_s;
float bitonic_sort_step_time;
float cudaMemcpy_host_to_device_time;
float cudaMemcpy_device_to_host_time;
int kernel_call_count = 0;
double bandwidth;
double numerator;
double denominator;

void bitonic_sort(float *values)
{
  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);

  hipMalloc((void**) &dev_values, size);
  
  // NEED TO TIME
  //MEM COPY FROM HOST TO DEVICE
  CALI_MARK_BEGIN(cudaMemcpy_host_to_device);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&cudaMemcpy_host_to_device_time, start, stop);

  CALI_MARK_END(cudaMemcpy_host_to_device);


  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  int j, k;

  // KERNEL BEING CALLED X NUMBER OF TIfMES HERE 
  /* Major step */
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      // NUMBER OF KERNEL CALLS HERE
      kernel_call_count += 1;
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&bitonic_sort_step_time, start, stop);
  // NEED TO TIME
  //MEM COPY FROM DEVICE TO HOST
  CALI_MARK_BEGIN(cudaMemcpy_device_to_host);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&cudaMemcpy_device_to_host_time, start, stop);

  hipDeviceSynchronize();

  CALI_MARK_END(cudaMemcpy_device_to_host);

  hipFree(dev_values);

  numerator = kernel_call_count * 6 * size / 1e9;
  denominator = bitonic_sort_step_time / 1000;

  bandwidth = numerator / denominator;


  printf("Kernel call count %d\n", kernel_call_count);
  printf("Numerator %f\n", numerator);
  printf("Denominator %f\n", denominator);
}


float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

int main(int argc, char *argv[])
{
    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;
    
    float *values = (float*) malloc( NUM_VALS * sizeof(float));
    array_fill(values, NUM_VALS);

    cali::ConfigManager mgr;
    mgr.start();

    // INSERT ADAIK CODE HERE
    
    mgr.stop();
    mgr.flush();
}