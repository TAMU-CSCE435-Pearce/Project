
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}