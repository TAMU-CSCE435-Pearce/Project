#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

#define OK 1
#define EXPECTATION_ERROR 1
#define MALLOC_ERROR 2
#define CUDA_ERROR 3

// source: https://github.com/chrishadi/cuda-sort/tree/main
// author: Chris Hadi
// I am using this source code for CUDA implementation of merge sort. I have added caliper and adiak annotations to the code.

const char* main_loop = "main loop";
const char* comm =  "comm";
const char* comm_large = "comm_large";
const char* small_comm1 = "small_comm1";
const char* small_comm2 = "small_comm2";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* data_init = "data_init";
const char* correctness = "correctness";


hipError_t mergeSortWithCuda(int* arr, unsigned int count);
bool assertArrEq(int* expected, int* actual, size_t size);
int testMergeSortWithCuda(int* actual, int* expected, const unsigned int count);

typedef struct mergeSortResult {
    hipError_t cudaStatus;
    char* msg;
} mergeSortResult_t;

__global__ void mergeSortKernel(int* arr, int* aux, unsigned int blockSize, const unsigned int last)
{
    int x = threadIdx.x;
    int start = blockSize * x;
    int end = start + blockSize - 1;
    int mid = start + (blockSize / 2) - 1;
    int l = start, r = mid + 1, i = start;

    if (end > last) { end = last; }
    if (start == end || end <= mid) { return; }

    while (l <= mid && r <= end) {
        if (arr[l] <= arr[r]) {
            aux[i++] = arr[l++];
        }
        else {
            aux[i++] = arr[r++];
        }
    }

    while (l <= mid) { aux[i++] = arr[l++]; }
    while (r <= end) { aux[i++] = arr[r++]; }

    for (i = start; i <= end; i++) {
        arr[i] = aux[i];
    }
}

inline mergeSortResult_t mergeSortError(hipError_t cudaStatus, char* msg) {
    mergeSortResult_t error;
    error.cudaStatus = cudaStatus;
    error.msg = msg;
    return error;
}

inline mergeSortResult_t mergeSortSuccess() {
    mergeSortResult_t success;
    success.cudaStatus = hipSuccess;
    return success;
}

inline mergeSortResult_t doMergeSortWithCuda(int* arr, unsigned int count, int* dev_arr, int* dev_aux) {
    const unsigned int last = count - 1;
    const unsigned size = count * sizeof(int);
    unsigned int threadCount;
    hipError_t cudaStatus;
    char msg[1024];

    // Copy input vectors from host memory to GPU buffers.
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);

    CALI_MARK_BEGIN(small_comm1);
    cudaStatus = hipMemcpy(dev_arr, arr, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        return mergeSortError(cudaStatus, "hipMemcpy failed!");
    }
    CALI_MARK_END(small_comm1);

    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    for (unsigned int blockSize = 2; blockSize < 2 * count; blockSize *= 2) {
        threadCount = count / blockSize;
        if (count % blockSize > 0) { threadCount++; }

        // Launch a kernel on the GPU with one thread for each block.
        mergeSortKernel<<<1, threadCount>>>(dev_arr, dev_aux, blockSize, last);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            sprintf(msg, "mergeSortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return mergeSortError(cudaStatus, msg);
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            sprintf(msg, "hipDeviceSynchronize returned error code %d after launching mergeSortKernel!\n", cudaStatus);
            return mergeSortError(cudaStatus, msg);
        }
    }
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);
    // Copy output vector from GPU buffer to host memory.
    CALI_MARK_BEGIN(small_comm2);
    cudaStatus = hipMemcpy(arr, dev_arr, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        return mergeSortError(cudaStatus, "hipMemcpy failed!");
    }
    CALI_MARK_END(small_comm2);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    return mergeSortSuccess();
}

hipError_t mergeSortWithCuda(int* arr, unsigned int count)
{
    const unsigned int size = count * sizeof(int);
    int* dev_arr = 0;
    int* dev_aux = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    // Allocate GPU buffers for two vectors (main and aux array).
    cudaStatus = hipMalloc((void**)&dev_arr, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_aux, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_arr);
        return cudaStatus;
    }

    mergeSortResult_t result = doMergeSortWithCuda(arr, count, dev_arr, dev_aux);

    if (result.cudaStatus != hipSuccess) {
        fprintf(stderr, result.msg);
    }

    hipFree(dev_arr);
    hipFree(dev_aux);

    return cudaStatus;
}

int main()
{
    CALI_CXX_MARK_FUNCTION;
    cali::ConfigManager mgr;
	mgr.start();
    CALI_MARK_BEGIN(main_loop);
    const unsigned int count = 64;
    const unsigned int size = count * sizeof(int);
    int status = MALLOC_ERROR;
    int* actual = (int*) malloc(size);
    int* expected = (int*) malloc(size);

    if (actual != NULL && expected != NULL) {
        status = testMergeSortWithCuda(actual, expected, count);
    }
    else {
        fprintf(stderr, "malloc failed!");
    }

    free(actual);
    free(expected);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    int cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return CUDA_ERROR;
    }
    CALI_MARK_END(main_loop);
    mgr.stop();
   	mgr.flush();
    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "Merge sort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "int"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", "4 bytes"); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", count); // The number of elements in input dataset (1000)
    adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    //adiak::value("num_procs", world_size); // The number of processors (MPI ranks)
    adiak::value("num_threads", count); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", count); // The number of CUDA blocks 
    adiak::value("group_num", "7"); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Online: https://github.com/chrishadi/cuda-sort/tree/main"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    return status;
}

int cmpInt(const void* a, const void* b) {
    return *(int*)a - *(int*)b;
}

int testMergeSortWithCuda(int* actual, int* expected, const unsigned int count) {
    CALI_MARK_BEGIN(data_init);
    for (unsigned int i = 0; i < count; i++) {
        expected[i] = actual[i] = rand();
    }
    CALI_MARK_END(data_init);

    qsort(expected, count, sizeof(int), cmpInt);

    hipError_t cudaStatus = mergeSortWithCuda(actual, count);
    CALI_MARK_BEGIN(correctness);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mergeSortWithCuda failed!");
        return CUDA_ERROR;
    }

    if (!assertArrEq(expected, actual, count * sizeof(int))) {
        puts("cuda sorted array is not equal to the qsorted array!");
        return EXPECTATION_ERROR;
    }
    puts("This is sorted by mergesort correctly");
    puts("test ok.");
    CALI_MARK_END(correctness);
    return OK;
    
}

bool assertArrEq(int* expected, int* actual, size_t size) {
    return memcmp(expected, actual, size) == 0;
}